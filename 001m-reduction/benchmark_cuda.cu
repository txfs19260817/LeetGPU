#include <hip/hip_runtime.h>
#include <nvbench/nvbench.cuh>
#include <random>
#include <vector>

extern "C" void solve(const float *input, float *output, int N);
extern "C" void solve1(const float* input, float* output, int N);
extern "C" void solve2(const float* input, float* output, int N);
extern "C" void solve3(const float* input, float* output, int N);
extern "C" void solve4(const float* input, float* output, int N);
extern "C" void solve5(const float* input, float* output, int N);
extern "C" void solve6(const float* input, float* output, int N);
extern "C" void solve_cub(const float* input, float* output, int N);

template <auto KernelFunc>
static void bench_reduction_impl(nvbench::state &state) {
  // Get the array size for this specific benchmark run
  const int N = static_cast<int>(state.get_int64("N"));

  // Generate random input data on the host
  std::vector<float> h_input(N);
  std::mt19937 rng(42); // Fixed seed for reproducibility
  std::uniform_real_distribution<float> dist(-1000.0f, 1000.0f);
  for (int i = 0; i < N; ++i) {
    h_input[i] = dist(rng);
  }

  // Allocate memory on the GPU
  float *d_input, *d_output;
  NVBENCH_CUDA_CALL(hipMalloc(&d_input, N * sizeof(float)));
  NVBENCH_CUDA_CALL(hipMalloc(&d_output, sizeof(float)));

  // Copy random input to GPU
  NVBENCH_CUDA_CALL(hipMemcpy(d_input, h_input.data(), N * sizeof(float),
                               hipMemcpyHostToDevice));
  NVBENCH_CUDA_CALL(hipMemset(d_output, 0, sizeof(float)));

  // state.exec() calls the specific KernelFunc that was passed in
  state.exec(nvbench::exec_tag::sync, [&](nvbench::launch &launch) {
    KernelFunc(d_input, d_output, N);
  });

  // Release GPU memory
  NVBENCH_CUDA_CALL(hipFree(d_input));
  NVBENCH_CUDA_CALL(hipFree(d_output));
}

static void bench_solve(nvbench::state &state) {
  bench_reduction_impl<solve>(state);
}
static void bench_solve_1(nvbench::state &state) {
    bench_reduction_impl<solve1>(state);
}
static void bench_solve_2(nvbench::state &state) {
    bench_reduction_impl<solve2>(state);
}
static void bench_solve_3(nvbench::state &state) {
    bench_reduction_impl<solve3>(state);
}
static void bench_solve_4(nvbench::state &state) {
    bench_reduction_impl<solve4>(state);
}
static void bench_solve_5(nvbench::state &state) {
    bench_reduction_impl<solve5>(state);
}
static void bench_solve_6(nvbench::state &state) {
    bench_reduction_impl<solve6>(state);
}
static void bench_solve_cub(nvbench::state &state) {
    bench_reduction_impl<solve_cub>(state);
}

// --- nvbench Benchmark Registration ---

const std::vector<nvbench::int64_t> ns = {
  10000,
  65535,
  1 << 16, // 65,536
  1 << 20, // 1,048,576 (1M)
  1 << 24, // 16,777,216 (16M)
  1 << 26, // 67,108,864 (64M)
  1 << 28, // 268,435,456 (256M)
  1 << 30  // 1,073,741,824 (1G)
};

NVBENCH_BENCH(bench_solve).set_name("reduction_baseline").add_int64_axis("N", ns);
NVBENCH_BENCH(bench_solve_1).set_name("reduction_v1").add_int64_axis("N", ns);
NVBENCH_BENCH(bench_solve_2).set_name("reduction_v2").add_int64_axis("N", ns);
NVBENCH_BENCH(bench_solve_3).set_name("reduction_v3").add_int64_axis("N", ns);
NVBENCH_BENCH(bench_solve_4).set_name("reduction_v4").add_int64_axis("N", ns);
NVBENCH_BENCH(bench_solve_5).set_name("reduction_v5").add_int64_axis("N", ns);
NVBENCH_BENCH(bench_solve_6).set_name("reduction_v6").add_int64_axis("N", ns);
NVBENCH_BENCH(bench_solve_cub).set_name("reduction_cub").add_int64_axis("N", ns);
