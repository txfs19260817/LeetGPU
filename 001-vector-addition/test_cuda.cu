#include <gtest/gtest.h>
#include <hip/hip_runtime.h>

// N = 1M elements
constexpr int N = 1 << 20;

extern "C" void solve(const float *A, const float *B, float *C, int N);
extern "C" void solve_stride(const float *A, const float *B, float *C, int N);

static void run_and_check(void (*solve_fn)(const float *, const float *, float *, int), int N)
{
    size_t size = N * sizeof(float);
    std::vector<float> h_A(N), h_B(N), h_C(N);

    for (int i = 0; i < N; i++)
    {
        h_A[i] = static_cast<float>(i);
        h_B[i] = static_cast<float>(i * 2);
    }

    float *d_A, *d_B, *d_C;
    hipMalloc(&d_A, size);
    hipMalloc(&d_B, size);
    hipMalloc(&d_C, size);

    hipMemcpy(d_A, h_A.data(), size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B.data(), size, hipMemcpyHostToDevice);

    // CUDA event profiling
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start);
    solve_fn(d_A, d_B, d_C, N);
    hipEventRecord(stop);
    hipEventSynchronize(stop);

    float ms = 0;
    hipEventElapsedTime(&ms, start, stop);

    hipMemcpy(h_C.data(), d_C, size, hipMemcpyDeviceToHost);

    // correctness check
    for (int i = 0; i < N; i++)
    {
        ASSERT_FLOAT_EQ(h_C[i], h_A[i] + h_B[i]);
    }

    std::cout << "Kernel execution time: " << ms << " ms" << std::endl;

    hipEventDestroy(start);
    hipEventDestroy(stop);
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
}

TEST(VectorAddTest, BasicKernel)
{
    run_and_check(solve, N);
}

TEST(VectorAddTest, StrideKernel)
{
    run_and_check(solve_stride, N);
}
