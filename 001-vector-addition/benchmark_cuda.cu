#include <benchmark/benchmark.h>
#include <hip/hip_runtime.h>

extern "C" void solve(const float *A, const float *B, float *C, int N);
extern "C" void solve_stride(const float *A, const float *B, float *C, int N);

static void BM_Solve(benchmark::State& state, void(*fn)(const float*, const float*, float*, int)) {
    int N = state.range(0);
    size_t size = N * sizeof(float);

    float *d_A, *d_B, *d_C;
    hipMalloc(&d_A, size);
    hipMalloc(&d_B, size);
    hipMalloc(&d_C, size);

    for (auto _ : state) {
        fn(d_A, d_B, d_C, N);
        hipDeviceSynchronize();
    }

    hipFree(d_A); hipFree(d_B); hipFree(d_C);
}

static void BM_SolveBasic(benchmark::State& state) { BM_Solve(state, solve); }
static void BM_SolveStride(benchmark::State& state) { BM_Solve(state, solve_stride); }

BENCHMARK(BM_SolveBasic)->Arg(1<<20);
BENCHMARK(BM_SolveStride)->Arg(1<<20);

BENCHMARK_MAIN();
