#include <hip/hip_runtime.h>

__global__ void vector_add(const float* A, const float* B, float* C, int N) {
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < N) {
        C[i] = A[i] + B[i];
    }
}

// Vector add with stride
__global__ void vector_add_stride(const float* A, const float* B, float* C, int N) {
    for (size_t i = blockIdx.x * blockDim.x + threadIdx.x;
         i < N;
         i += blockDim.x * gridDim.x) {
        C[i] = A[i] + B[i];
    }
}

// A, B, C are device pointers (i.e. pointers to memory on the GPU)
extern "C" void solve(const float* A, const float* B, float* C, int N) {
    int threadsPerBlock = 256;
    int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;

    vector_add<<<blocksPerGrid, threadsPerBlock>>>(A, B, C, N);
    hipDeviceSynchronize();
}

extern "C" void solve_stride(const float* A, const float* B, float* C, int N) {
    int threadsPerBlock = 256;
    int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;

    vector_add_stride<<<blocksPerGrid, threadsPerBlock>>>(A, B, C, N);
    hipDeviceSynchronize();
}